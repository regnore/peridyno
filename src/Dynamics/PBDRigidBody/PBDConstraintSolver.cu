#include "hip/hip_runtime.h"
#include "PBDConstraintSolver.h"

namespace dyno
{
	IMPLEMENT_TCLASS(PBDConstraintSolver, TDataType)

		template<typename TDataType>
	PBDConstraintSolver<TDataType>::PBDConstraintSolver()
		: ConstraintModule()
	{
		this->inContacts()->tagOptional(true);
		this->inJoint()->tagOptional(true);
	}

	template<typename TDataType>
	PBDConstraintSolver<TDataType>::~PBDConstraintSolver()
	{
	}

	template <typename Coord, typename Real>
	__global__ void PBDRB_UpdateXV(
		DArray<Coord> x,
		DArray<Coord> x_prev,
		DArray<Coord> v,
		DArray<Real> mass,
		Coord a_ext,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= x.size()) return;

		x_prev[pId] = x[pId];
		v[pId] += h * a_ext;
		x[pId] += v[pId] * h;
	}

	template <typename Coord, typename Matrix, typename Quat, typename Real>
	__global__ void PBDRB_UpdateQW(
		DArray<Quat> q,
		DArray<Quat> q_prev,
		DArray<Matrix> R,
		DArray<Matrix> I,
		DArray<Coord> w,
		DArray<Matrix> I_init,
		Coord tau_ext,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= q.size()) return;

		q_prev[pId] = q[pId];
		w[pId] += h * I[pId].inverse() * (tau_ext - (w[pId].cross(I[pId] * w[pId])));
		q[pId] += h * 0.5f * (Quat(w[pId][0], w[pId][1], w[pId][2], 0.0) * q[pId]);
		q[pId] = q[pId].normalize();

		R[pId] = q[pId].toMatrix3x3();
		I[pId] = R[pId] * I_init[pId] * R[pId].inverse();
	}

	template <typename Coord, typename Quat, typename Real>
	__global__ void PBDRB_CalcVW(
		DArray<Coord> x,
		DArray<Coord> x_prev,
		DArray<Coord> v,
		DArray<Coord> v_prev,
		DArray<Quat> q,
		DArray<Quat> q_prev,
		DArray<Coord> w,
		DArray<Coord> w_prev,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= x.size()) return;

		v_prev[pId] = v[pId];
		w_prev[pId] = w[pId];
		v[pId] = (x[pId] - x_prev[pId]) / h;

		Quat dq = q[pId] * (q_prev[pId].inverse());
		w[pId] = (dq.w >= 0 ? 1 : -1) * 2 * Coord(dq.x, dq.y, dq.z) / h;
	}

	template <typename Matrix, typename Quat>
	__global__ void PBDRB_UpdateRI(
		DArray<Quat> q,
		DArray<Matrix> I_init,
		DArray<Matrix> I,
		DArray<Matrix> R)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= q.size()) return;

		R[pId] = q[pId].toMatrix3x3();
		I[pId] = R[pId] * I_init[pId] * R[pId].inverse();
	}

	template <typename Coord, typename Quat, typename Matrix, typename Real, typename ContactPair>
	__global__ void PBDRB_SolvePositions(
		DArray<Coord> x,
		DArray<Quat> q,
		DArray<Real> m,
		DArray<Matrix> I,
		DArray<Real> lambdaN,
		DArray<Real> alpha,
		DArray<ContactPair> nbq,
		DArray<Real> stepInv,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= nbq.size()) return;

		int idx1 = nbq[pId].bodyId1;
		int idx2 = nbq[pId].bodyId2;
		Coord n = -nbq[pId].normal1;
		n /= n.norm();
		Real c = nbq[pId].interpenetration;
		Real tildeAlpha = alpha[pId] / h / h;

		if (c > 0)
		{
			if (idx2 != -1)
			{
				Coord r1 = nbq[pId].pos1 - x[idx1];
				Coord r2 = nbq[pId].pos1 - x[idx2];

				Coord temp3 = r1.cross(n);
				Coord temp4 = r2.cross(n);

				Real w1 = 1.0f / m[idx1] + (Real)(temp3.dot(I[idx1].inverse() * temp3));
				Real w2 = 1.0f / m[idx2] + (Real)(temp4.dot(I[idx2].inverse() * temp4));

				Real dLambdaN = ((-c - tildeAlpha * lambdaN[pId]) / (w1 + w2 + tildeAlpha));
				dLambdaN/= (stepInv[idx1] + stepInv[idx2]);
				lambdaN[pId] += dLambdaN;

				Coord p = dLambdaN * n;

				Coord temp1 = I[idx1].inverse() * (r1.cross(p));
				Coord temp2 = I[idx2].inverse() * (r2.cross(p));
				Quat temp5 = Quat(temp1[0], temp1[1], temp1[2], 0) * q[idx1] * 0.5f;
				Quat temp6 = Quat(temp2[0], temp2[1], temp2[2], 0) * q[idx2] * 0.5f;

				atomicAdd(&x[idx1][0], p[0] / m[idx1]);
				atomicAdd(&x[idx1][1], p[1] / m[idx1]);
				atomicAdd(&x[idx1][2], p[2] / m[idx1]);

				atomicAdd(&x[idx2][0], -p[0] / m[idx2]);
				atomicAdd(&x[idx2][1], -p[1] / m[idx2]);
				atomicAdd(&x[idx2][2], -p[2] / m[idx2]);

				atomicAdd(&q[idx1].x, temp5.x);
				atomicAdd(&q[idx1].y, temp5.y);
				atomicAdd(&q[idx1].z, temp5.z);
				atomicAdd(&q[idx1].w, temp5.w);

				atomicAdd(&q[idx2].x, -temp6.x);
				atomicAdd(&q[idx2].y, -temp6.y);
				atomicAdd(&q[idx2].z, -temp6.z);
				atomicAdd(&q[idx2].w, -temp6.w);
			}
			else
			{
				Coord r1 = nbq[pId].pos1 - x[idx1];
				Coord temp3 = r1.cross(n);

				Real w1 = 1.0f / m[idx1] + (Real)(temp3.dot(I[idx1].inverse() * temp3));
				Real w2 = 0.0f;

				//printf("%.10f\t%.10f\n",1/w1,m[idx1]);
				Real dLambdaN = ((-c - tildeAlpha * lambdaN[pId]) / (w1 + w2 + tildeAlpha));
				dLambdaN /= stepInv[idx1];
				lambdaN[pId] += dLambdaN;
				Coord p = dLambdaN * n;
				Coord temp1 = I[idx1].inverse() * (r1.cross(p));
				Quat temp2 = 0.5f * Quat(temp1[0], temp1[1], temp1[2], 0) * q[idx1];
				atomicAdd(&x[idx1][0], p[0] / m[idx1]);
				atomicAdd(&x[idx1][1], p[1] / m[idx1]);
				atomicAdd(&x[idx1][2], p[2] / m[idx1]);

				atomicAdd(&q[idx1].x, temp2.x);
				atomicAdd(&q[idx1].y, temp2.y);
				atomicAdd(&q[idx1].z, temp2.z);
				atomicAdd(&q[idx1].w, temp2.w);
			}
		}
	}

	template <typename Coord, typename Quat, typename Matrix, typename Real, typename ContactPair>
	__global__ void PBDRB_SolvePositionsFriction(
		DArray<Coord> x,
		DArray<Coord> x_prev,
		DArray<Quat> q,
		DArray<Quat> q_prev,
		DArray<Real> m,
		DArray<Matrix> I,
		DArray<Real> lambdaN,
		DArray<Real> lambdaT,
		DArray<Real> alpha,
		DArray<Real> miuS,
		DArray<ContactPair> nbq,
		DArray<Real> stepInv,
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= nbq.size()) return;

		int idx1 = nbq[pId].bodyId1;
		int idx2 = nbq[pId].bodyId2;
		Coord n = -nbq[pId].normal1;
		n /= n.norm();
		Real tildeAlpha = alpha[pId] / h / h;

		if (idx2 != -1)
		{
			Coord r1 = nbq[pId].pos1 - x[idx1];
			Coord r2 = nbq[pId].pos1 - x[idx2];

			Coord p1 = x[idx1] + r1;
			Coord p2 = x[idx2] + r2;

			Coord p1Bar = x_prev[idx1] + (q_prev[idx1] * q[idx1].inverse()).normalize().rotate(r1);
			Coord p2Bar = x_prev[idx2] + (q_prev[idx2] * q[idx2].inverse()).normalize().rotate(r2);

			Coord dP = (p1 - p1Bar) - (p2 - p2Bar);
			Coord dP_t = dP - (dP.dot(n)) * n;

			Coord temp3 = r1.cross(n);
			Coord temp4 = r2.cross(n);

			Real w1 = 1.0f / m[idx1] + (Real)(temp3.dot(I[idx1].inverse() * temp3));
			Real w2 = 1.0f / m[idx2] + (Real)(temp4.dot(I[idx2].inverse() * temp4));

			Real dLambdaT = ((-dP_t.norm() - tildeAlpha * lambdaT[pId]) / (w1 + w2 + tildeAlpha));
			dLambdaT /= stepInv[idx1] + stepInv[idx2];
			lambdaT[pId] += dLambdaT;

			if (dP_t.norm()> EPSILON && lambdaT[pId] > (miuS[idx1] + miuS[idx2]) * lambdaN[pId])
			{
				Coord p = dLambdaT * (dP_t / dP_t.norm());
				Coord temp7 = I[idx1].inverse() * (r1.cross(p));
				Coord temp8 = I[idx2].inverse() * (r2.cross(p));
				Quat temp9 = Quat(temp7[0], temp7[1], temp7[2], 0) * q[idx1] * 0.5f;
				Quat temp10 = Quat(temp8[0], temp8[1], temp8[2], 0) * q[idx2] * 0.5f;

				atomicAdd(&x[idx1][0], p[0] / m[idx1]);
				atomicAdd(&x[idx1][1], p[1] / m[idx1]);
				atomicAdd(&x[idx1][2], p[2] / m[idx1]);

				atomicAdd(&x[idx2][0], -p[0] / m[idx2]);
				atomicAdd(&x[idx2][1], -p[1] / m[idx2]);
				atomicAdd(&x[idx2][2], -p[2] / m[idx2]);

				atomicAdd(&q[idx1].x, temp9.x);
				atomicAdd(&q[idx1].y, temp9.y);
				atomicAdd(&q[idx1].z, temp9.z);
				atomicAdd(&q[idx1].w, temp9.w);

				atomicAdd(&q[idx2].x, -temp10.x);
				atomicAdd(&q[idx2].y, -temp10.y);
				atomicAdd(&q[idx2].z, -temp10.z);
				atomicAdd(&q[idx2].w, -temp10.w);
			}
		}
			else
			{
				Coord r1 = nbq[pId].pos1 - x[idx1];
				Coord p1 = x[idx1] + r1;
				Coord p1Bar = x_prev[idx1] + (q_prev[idx1] * q[idx1].inverse()).normalize().rotate(r1);

				Coord temp3 = r1.cross(n);

				Real w1 = 1.0f / m[idx1] + (Real)(temp3.dot(I[idx1].inverse() * temp3));
				Real w2 =0.0f;

				Coord dP = (p1 - p1Bar);
				Coord dP_t = dP - (dP.dot(n)) * n;
				Real dLambdaT = ((-dP_t.norm() - tildeAlpha * lambdaT[pId]) / (w1 + w2 + tildeAlpha));
				dLambdaT /= stepInv[idx1];
				//printf("%.10f\t%.10f\t%.10f\t%.10f\t%.10f\n",dP.norm(),dP_t.norm(),dLambdaT,dP_t.dot(n),lambdaN[pId]);
				lambdaT[pId] += dLambdaT;

				if (dP_t.norm()>EPSILON && lambdaT[pId] > miuS[idx1] * lambdaN[pId])
				{
					Coord p = dLambdaT * (dP_t/ dP_t.norm());
					Coord temp7 = I[idx1].inverse() * (r1.cross(p));
					Quat temp9 = Quat(temp7[0], temp7[1], temp7[2], 0) * q[idx1] * 0.5f;

					//printf("%.10f\n",);

					atomicAdd(&x[idx1][0], p[0] / m[idx1]);
					atomicAdd(&x[idx1][1], p[1] / m[idx1]);
					atomicAdd(&x[idx1][2], p[2] / m[idx1]);

					atomicAdd(&q[idx1].x, temp9.x);
					atomicAdd(&q[idx1].y, temp9.y);
					atomicAdd(&q[idx1].z, temp9.z);
					atomicAdd(&q[idx1].w, temp9.w);
				}
			}
		}



		template <typename Coord, typename Quat, typename Matrix, typename Real, typename Joint>
		__global__ void PBDRB_SolveJointPosition(
			DArray<Coord> x,
			DArray<Quat> q,
			DArray<Real> m,
			DArray<Matrix> I,
			DArray<Joint> joint,
			DArray<Real> lambdaJ,
			DArray<Real> jCnt,
			Real h)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= joint.size()) return;

			int idx1 = joint[pId].bodyId1;
			int idx2 = joint[pId].bodyId2;

			Coord r1 = q[idx1].normalize().rotate(joint[pId].offset1);
			Coord r2 = q[idx2].normalize().rotate(joint[pId].offset2);
			Coord n = -(x[idx2] + r2) + (x[idx1] + r1);
			Real c = n.norm();
			n /= n.norm();

			Real tildeAlpha = joint[pId].alpha / h / h;

			if (c > 0)
			{
				Coord temp1 = r1.cross(n);
				Coord temp2 = r2.cross(n);

				Real w1 = 1.0f / m[idx1] + (Real)(temp1.dot(I[idx1].inverse() * temp1));
				Real w2 = 1.0f / m[idx2] + (Real)(temp2.dot(I[idx2].inverse() * temp2));
				Real dLambdaJ = ((-c - tildeAlpha * lambdaJ[pId]) / (w1 + w2 + tildeAlpha));
				lambdaJ[pId] += dLambdaJ;

				Coord p = dLambdaJ * n;

				Coord temp3 = I[idx1].inverse() * (r1.cross(p));
				Coord temp4 = I[idx2].inverse() * (r2.cross(p));
				Quat temp5 = Quat(temp3[0], temp3[1], temp3[2], 0) * q[idx1] * 0.5f;
				Quat temp6 = Quat(temp4[0], temp4[1], temp4[2], 0) * q[idx2] * 0.5f;

				//printf("%f",jCnt[idx1]);

				atomicAdd(&x[idx1][0], p[0] / m[idx1] / jCnt[idx1]);
				atomicAdd(&x[idx1][1], p[1] / m[idx1] / jCnt[idx1]);
				atomicAdd(&x[idx1][2], p[2] / m[idx1] / jCnt[idx1]);

				atomicAdd(&x[idx2][0], -p[0] / m[idx2] / jCnt[idx2]);
				atomicAdd(&x[idx2][1], -p[1] / m[idx2] / jCnt[idx2]);
				atomicAdd(&x[idx2][2], -p[2] / m[idx2] / jCnt[idx2]);

				atomicAdd(&q[idx1].x, temp5.x / jCnt[idx1]);
				atomicAdd(&q[idx1].y, temp5.y / jCnt[idx1]);
				atomicAdd(&q[idx1].z, temp5.z / jCnt[idx1]);
				atomicAdd(&q[idx1].w, temp5.w / jCnt[idx1]);

				atomicAdd(&q[idx2].x, -temp6.x / jCnt[idx2]);
				atomicAdd(&q[idx2].y, -temp6.y / jCnt[idx2]);
				atomicAdd(&q[idx2].z, -temp6.z / jCnt[idx2]);
				atomicAdd(&q[idx2].w, -temp6.w / jCnt[idx2]);
			}
		}

		template <typename Coord, typename Quat, typename Matrix, typename Real, typename Joint>
		__global__ void PBDRB_SolveJointAngle(
			DArray<Coord> x,
			DArray<Quat> q,
			DArray<Real> m,
			DArray<Coord> a0,
			DArray<Matrix> I,
			DArray<Joint> joint,
			DArray<Real> lambdaJA,
			DArray<Real> jCnt,
			Real h)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= joint.size()) return;

			int idx1 = joint[pId].bodyId1;
			int idx2 = joint[pId].bodyId2;
			Coord a1 = q[idx1].normalize().rotate(a0[idx1]);
			Coord a2 = q[idx2].normalize().rotate(a0[idx2]);
			Coord dq_h = a1.cross(a2);
			Real theta = asinf(dq_h.norm());
			if (theta != 0.0f)
			{
				Coord n = dq_h / dq_h.norm();

				Real tildeAlpha = joint[pId].alpha / h / h;

				Real w1 = n.dot(I[idx1].inverse() * n);
				Real w2 = n.dot(I[idx2].inverse() * n);
				Real dLambdaJA = ((- theta - tildeAlpha * lambdaJA[pId]) / (w1 + w2 + tildeAlpha));
				lambdaJA[pId] += dLambdaJA;
				//printf("%.10f\t%.10f\t%.10f\t%.10f\t%.10f\t%.10f\n", dLambdaJA,theta/M_PI*180.0f,dq_h.norm(),a1.norm(),a2.norm(),w1+w2);

				Coord p = -dLambdaJA * n;

				Coord temp1 = I[idx1].inverse() * p;
				Coord temp2 = I[idx2].inverse() * p;
				Quat temp3 = Quat(temp1[0], temp1[1], temp1[2], 0) * q[idx1] * 0.5f;
				Quat temp4 = Quat(temp2[0], temp2[1], temp2[2], 0) * q[idx2] * 0.5f;

				atomicAdd(&q[idx1].x, temp3.x / jCnt[idx1]);
				atomicAdd(&q[idx1].y, temp3.y / jCnt[idx1]);
				atomicAdd(&q[idx1].z, temp3.z / jCnt[idx1]);
				atomicAdd(&q[idx1].w, temp3.w / jCnt[idx1]);

				atomicAdd(&q[idx2].x, -temp4.x / jCnt[idx2]);
				atomicAdd(&q[idx2].y, -temp4.y / jCnt[idx2]);
				atomicAdd(&q[idx2].z, -temp4.z / jCnt[idx2]);
				atomicAdd(&q[idx2].w, -temp4.w / jCnt[idx2]);
			}
		}

		template <typename Coord, typename Matrix, typename Real, typename ContactPair>
		__global__ void PBDRB_SolveVelocities(
			DArray<Coord> x,
			DArray<Coord> v,
			DArray<Coord> v_prev,
			DArray<Coord> w,
			DArray<Coord> w_prev,
			DArray<Real> m,
			DArray<Matrix> I,
			DArray<Real> lambda,
			DArray<Real> miu,
			DArray<ContactPair> nbq,
			DArray<Real> stepInv,
			Real restituteCoef,
			Real h)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= nbq.size()) return;

			int idx1 = nbq[pId].bodyId1;
			int idx2 = nbq[pId].bodyId2;
			Coord n = -nbq[pId].normal1;
			n /= n.norm();

			if (idx2 != -1)
			{
				Coord r1 = nbq[pId].pos1 - x[idx1];
				Coord r2 = nbq[pId].pos1 - x[idx2];

				Coord vv = (v[idx1] + w[idx1].cross(r1)) - (v[idx2] + w[idx2].cross(r2));
				Real v_n = n.dot(vv);
				Coord v_t = vv - v_n * n;

				Coord vv_prev = (v_prev[idx1] + w_prev[idx1].cross(r1)) - (v_prev[idx2] + w_prev[idx2].cross(r2));
				Real v_n_prev = n.dot(vv_prev) > 0 ? 0.0f : abs(n.dot(vv_prev));

				Coord dv = Coord(0.0f);
				if (abs(v_n) < h * 9.8f * 2)
					dv += n * (-v_n);
				else
					dv += n * (-v_n - restituteCoef * v_n_prev);

				if (v_t.norm() > 0)
				{
					Real miu_d = (miu[idx1] + miu[idx2]) / 2;
					if (v_t.norm() < EPSILON)
						dv += -v_t;
					else
						dv += -(v_t / v_t.norm() * min(miu_d * (abs(lambda[pId]) * (stepInv[idx1] + stepInv[idx2]) * (stepInv[idx1] + stepInv[idx2]) / h), v_t.norm()));

					Coord temp1 = r1.cross(n);
					Coord temp2 = r2.cross(n);

					Real w1 = 1.0f / m[idx1] + (Real)(temp1.dot(I[idx1].inverse() * temp1));
					Real w2 = 1.0f / m[idx2] + (Real)(temp2.dot(I[idx2].inverse() * temp2));
					Coord p = dv / (w1 + w2) / (stepInv[idx1] + stepInv[idx2]);

					Coord temp3 = I[idx1].inverse() * (r1.cross(p));
					Coord temp4 = I[idx2].inverse() * (r2.cross(p));

					atomicAdd(&v[idx1][0], p[0] / m[idx1]);
					atomicAdd(&v[idx1][1], p[1] / m[idx1]);
					atomicAdd(&v[idx1][2], p[2] / m[idx1]);

					atomicAdd(&v[idx2][0], -p[0] / m[idx2]);
					atomicAdd(&v[idx2][1], -p[1] / m[idx2]);
					atomicAdd(&v[idx2][2], -p[2] / m[idx2]);

					atomicAdd(&w[idx1][0], temp3[0]);
					atomicAdd(&w[idx1][1], temp3[1]);
					atomicAdd(&w[idx1][2], temp3[2]);

					atomicAdd(&w[idx2][0], -temp4[0]);
					atomicAdd(&w[idx2][1], -temp4[1]);
					atomicAdd(&w[idx2][2], -temp4[2]);
				}
			}
			else
			{
				Coord r1 = nbq[pId].pos1 - x[idx1];

				Coord vv = v[idx1] + w[idx1].cross(r1);
				Real v_n = n.dot(vv);
				Coord v_t = vv - v_n * n;

				Coord vv_prev = v_prev[idx1] + w_prev[idx1].cross(r1);
				Real v_n_prev = n.dot(vv_prev) > 0 ? 0.0f : abs(n.dot(vv_prev));

				Coord dv = Coord(0.0f);
				if (abs(v_n) < h * 9.8f * 2)
					dv += n * (-v_n);
				else
					dv += n * (-v_n - restituteCoef * v_n_prev);


				//printf("%.10f\t%.10f\n", v_n, v_n_prev);

				if (v_t.norm() > 0)
				{
					Real miu_d = miu[idx1];
					if (v_t.norm() < EPSILON)
						dv += -v_t;
					else
						dv += -(v_t / v_t.norm() * min(miu_d * (abs(lambda[pId])* stepInv[idx1] * stepInv[idx1] / h ), v_t.norm()));

					//printf("%.10f\t%.10f\n", miu_d * (abs(lambda[pId]) * stepInv[idx1] * stepInv[idx1] / h), v_t.norm());
					Coord temp1 = r1.cross(n);

					Real w1 = 1.0f / m[idx1] + temp1.dot(I[idx1].inverse() * temp1);
					Real w2 = 0.0f;
					Coord p = dv / (w1 + w2) / stepInv[idx1];

					Coord temp3 = I[idx1].inverse() * (r1.cross(p));

					atomicAdd(&v[idx1][0], p[0] / m[idx1]);
					atomicAdd(&v[idx1][1], p[1] / m[idx1]);
					atomicAdd(&v[idx1][2], p[2] / m[idx1]);
					//printf("%.10f\t%.10f\t%.10f\n", v[idx1][0], v[idx1][1], v[idx1][2]);

					atomicAdd(&w[idx1][0], temp3[0]);
					atomicAdd(&w[idx1][1], temp3[1]);
					atomicAdd(&w[idx1][2], temp3[2]);
				}
			}
		}

		template<typename TDataType>
		void PBDConstraintSolver<TDataType>::constrain()
		{
			uint num = this->inCenter()->size();
			Real dt = this->inTimeStep()->getData();
			uint numSubsteps = this->varNumSubsteps()->getData();
			Real h = dt / numSubsteps;

			//printf("%d\n",this->inJoint()->size());

			if (this->x_prev.size() == 0)
				this->x_prev.resize(num);

			if (this->q_prev.size() == 0)
				this->q_prev.resize(num);

			if (this->v_prev.size() == 0)
				this->v_prev.resize(num);

			if (this->w_prev.size() == 0)
				this->w_prev.resize(num);

			Coord g = Coord(0.0f, -9.8f, 0.0f);
			Coord tau = Coord(0.0f, 0.0f, 0.0f);
			cuExecute(num,
				PBDRB_UpdateXV,
				this->inCenter()->getData(),
				x_prev,
				this->inVelocity()->getData(),
				this->inMass()->getData(),
				g,
				h);

			cuExecute(num,
				PBDRB_UpdateQW,
				this->inQuaternion()->getData(),
				q_prev,
				this->inRotationMatrix()->getData(),
				this->inInertia()->getData(),
				this->inAngularVelocity()->getData(),
				this->inInitialInertia()->getData(),
				tau,
				h);

			uint numC = 0;
			uint numJ = 0;
			if (this->inContacts()->size() > 0 || this->inJoint()->size() > 0)
			{
				this->initialize();
				numC = this->inContacts()->size();
				numJ = this->inJoint()->size();
				this->mLambdaJ.resize(numJ);
				this->mLambdaJ.reset();
				this->mLambdaJA.resize(numJ);
				this->mLambdaJA.reset();
				this->mLambdaT.resize(numC);
				this->mLambdaT.reset();
				this->mLambdaN.resize(numC);
				this->mLambdaN.reset();
			}

			if (numJ > 0)
			{
				cuExecute(numJ,
					PBDRB_SolveJointPosition,
					this->inCenter()->getData(),
					this->inQuaternion()->getData(),
					this->inMass()->getData(),
					this->inInertia()->getData(),
					this->inJoint()->getData(),
					this->mLambdaJ,
					this->mJointNumber,
					h);

				cuExecute(
					num,
					PBDRB_UpdateRI,
					this->inQuaternion()->getData(),
					this->inInitialInertia()->getData(),
					this->inInertia()->getData(),
					this->inRotationMatrix()->getData());

				cuExecute(numJ,
					PBDRB_SolveJointAngle,
					this->inCenter()->getData(),
					this->inQuaternion()->getData(),
					this->inMass()->getData(), 
					this->inA()->getData(),
					this->inInertia()->getData(),
					this->inJoint()->getData(),
					this->mLambdaJ,
					this->mJointNumber,
					h);

				cuExecute(
					num,
					PBDRB_UpdateRI,
					this->inQuaternion()->getData(),
					this->inInitialInertia()->getData(),
					this->inInertia()->getData(),
					this->inRotationMatrix()->getData());
			}

			if (numC > 0)
			{
				cuExecute(numC,
					PBDRB_SolvePositions,
					this->inCenter()->getData(),
					this->inQuaternion()->getData(),
					this->inMass()->getData(),
					this->inInertia()->getData(),
					this->mLambdaN,
					this->mAlpha,
					this->mAllConstraints,
					this->mContactNumber,
					h);


				cuExecute(
					num,
					PBDRB_UpdateRI,
					this->inQuaternion()->getData(),
					this->inInitialInertia()->getData(),
					this->inInertia()->getData(),
					this->inRotationMatrix()->getData());

				if (this->varStaticFrictionEnabled()->getData())
				{

					cuExecute(numC,
						PBDRB_SolvePositionsFriction,
						this->inCenter()->getData(),
						this->x_prev,
						this->inQuaternion()->getData(),
						this->q_prev,
						this->inMass()->getData(),
						this->inInertia()->getData(),
						this->mLambdaN,
						this->mLambdaT,
						this->mAlpha,
						this->inStaticFriction()->getData(),
						this->mAllConstraints,
						this->mContactNumber,
						h);

					cuExecute(
						num,
						PBDRB_UpdateRI,
						this->inQuaternion()->getData(),
						this->inInitialInertia()->getData(),
						this->inInertia()->getData(),
						this->inRotationMatrix()->getData());
				}
			}

			cuExecute(
				num,
				PBDRB_CalcVW,
				this->inCenter()->getData(),
				x_prev,
				this->inVelocity()->getData(),
				this->v_prev,
				this->inQuaternion()->getData(),
				q_prev,
				this->inAngularVelocity()->getData(),
				this->w_prev,
				h);

			if (this->varDynamicFrictionEnabled()->getData())
			{
				if (numC > 0)
				{
					cuExecute(
						numC,
						PBDRB_SolveVelocities,
						this->inCenter()->getData(),
						this->inVelocity()->getData(),
						this->v_prev,
						this->inAngularVelocity()->getData(),
						this->w_prev,
						this->inMass()->getData(),
						this->inInertia()->getData(),
						this->mLambdaN,
						this->inDynamicFriction()->getData(),
						this->mAllConstraints,
						this->mContactNumber,
						this->varRestituteCoef()->getData(),
						h);
				}
			}

			this->mLambdaN.reset();
			this->mLambdaT.reset();
			this->mLambdaJ.reset();
			this->mLambdaJA.reset();
			this->mAllConstraints.reset();
		}

		template <typename ContactPair>
		__global__ void CalculateNbrCons(
			DArray<ContactPair> nbc,
			DArray<Real> nbrCnt)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= nbc.size()) return;

			int idx1 = nbc[pId].bodyId1;
			int idx2 = nbc[pId].bodyId2;

			if (idx1 != -1)
				atomicAdd(&nbrCnt[idx1], 1.0f);
			if (idx2 != -1)
				atomicAdd(&nbrCnt[idx2], 1.0f);
		}

		template <typename Real, typename Joint>
		__global__ void CalculateJointCons(
			DArray<Joint> j,
			DArray<Real> jCnt)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= j.size()) return;

			int idx1 = j[pId].bodyId1;
			int idx2 = j[pId].bodyId2;

			if (idx1 != -1)
				atomicAdd(&jCnt[idx1], 1.0f);
			if (idx2 != -1)
				atomicAdd(&jCnt[idx2], 1.0f);
		}

		template<typename TDataType>
		void PBDConstraintSolver<TDataType>::initialize()
		{
			if (this->inContacts()->isEmpty()&& this->inJoint()->isEmpty())
				return;

			int sizeOfContacts = this->inContacts()->size();
			int sizeOfConstraints = sizeOfContacts;

			int sizeOfJoints = this->inJoint()->size();

			mAllConstraints.resize(sizeOfConstraints);

			if (sizeOfContacts > 0)
			{
				auto& contacts = this->inContacts()->getData();
				mAllConstraints.assign(contacts, contacts.size(), 0, 0);
			}

			mLambdaN.resize(sizeOfConstraints);
			mLambdaT.resize(sizeOfConstraints);
			mLambdaJ.resize(sizeOfJoints);
			mLambdaJA.resize(sizeOfJoints);
			mAlpha.resize(sizeOfConstraints);

			auto sizeOfRigids = this->inCenter()->size();
			mContactNumber.resize(sizeOfRigids);
			mJointNumber.resize(sizeOfRigids);

			mLambdaN.reset();
			mLambdaT.reset();
			mLambdaJ.reset();
			mLambdaJA.reset();
			mContactNumber.reset();
			mJointNumber.reset();
			mAlpha.reset();

			if (sizeOfConstraints > 0)
			{
				cuExecute(sizeOfConstraints,
					CalculateNbrCons,
					mAllConstraints,
					mContactNumber
				);
			}
			if (sizeOfJoints > 0) 
			{
				cuExecute(sizeOfJoints,
					CalculateJointCons,
					this->inJoint()->getData(),
					mJointNumber
				);
			}
		}

		DEFINE_CLASS(PBDConstraintSolver);
	
}