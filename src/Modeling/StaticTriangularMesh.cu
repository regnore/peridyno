#include "hip/hip_runtime.h"
#include "StaticTriangularMesh.h"

#include "Topology/TriangleSet.h"

namespace dyno
{
	IMPLEMENT_TCLASS(StaticTriangularMesh, TDataType)

	template<typename TDataType>
	StaticTriangularMesh<TDataType>::StaticTriangularMesh()
		: Node()
	{
		auto triSet = std::make_shared<TriangleSet<TDataType>>();
		this->stateTopology()->setDataPtr(triSet);

		this->outTriangleSet()->setDataPtr(triSet);
	}

	template<typename TDataType>
	void StaticTriangularMesh<TDataType>::resetStates()
	{
		auto triSet = TypeInfo::cast<TriangleSet<TDataType>>(this->stateTopology()->getDataPtr());

		if (this->varFileName()->getDataPtr()->string() == "")
			return;

		triSet->loadObjFile(this->varFileName()->getDataPtr()->string());

		triSet->scale(this->varScale()->getData());
		triSet->translate(this->varLocation()->getData());

		Node::resetStates();

		initPos.resize(triSet->getPoints().size());
		initPos.assign(triSet->getPoints());
		center = this->varCenter()->getData();
		centerInit = center;

	}

	template <typename Coord, typename Matrix>
	__global__ void K_InitKernelFunctionMesh(
		DArray<Coord> posArr,
		DArray<Coord> posInit,
		Coord center,
		Coord centerInit,
		Matrix rotation
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size())
			return;
		Coord pos;
		pos = posInit[pId] - centerInit;
		pos = rotation * pos;
		posArr[pId] = pos + center;

	}


	template<typename TDataType>
	void StaticTriangularMesh<TDataType>::updateStates()
	{
		//printf("update static boundary\n");

		Coord velocity = this->varVelocity()->getData();
		Coord angularVelocity = this->varAngularVelocity()->getData();

		//printf("velocity = %.10lf %.10lf %.10lf\n", velocity[0], velocity[1], velocity[2]);

		Real dt = 0.001f;
		rotQuat = rotQuat.normalize();
		rotQuat += dt * 0.5f *
			Quat<Real>(angularVelocity[0], angularVelocity[1], angularVelocity[2], 0.0)*(rotQuat);

		rotQuat = rotQuat.normalize();
		rotMat = rotQuat.toMatrix3x3();

		center += velocity * dt;

		auto triSet = TypeInfo::cast<TriangleSet<TDataType>>(this->stateTopology()->getDataPtr());

		cuExecute(triSet->getPoints().size(),
			K_InitKernelFunctionMesh,
			triSet->getPoints(),
			initPos,
			center,
			centerInit,
			rotMat
			);

	}

	DEFINE_CLASS(StaticTriangularMesh);
}